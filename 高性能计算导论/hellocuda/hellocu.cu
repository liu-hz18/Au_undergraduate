
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"//���������cuBLAS

//���ļ�������GPU��������
//ע�⽫���ļ���ӵ� Դ�ļ��У� ��������Դ�ļ�
//����Ŀ�����Կ���ͨ�� ��cuda_debug_x64_PropertySheet�����

// defining
void nvcc_test(){
	std::cout << "I have been processed by NVCC!" << std::endl;
}

//��ӡGPU�豸��Ϣ
void print_GPU_device_info(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i<deviceCount; i++){
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		std::cout << "ʹ��GPU device " << i << ": " << devProp.name << std::endl;
		std::cout << "�豸ȫ���ڴ������� " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
		std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
		std::cout << "ÿ��SM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
		std::cout << "ÿ��SM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
		std::cout << "ÿ��Block�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
		std::cout << "ÿ��Block������߳�����" << devProp.maxThreadsPerBlock << std::endl;
		std::cout << "ÿ��Block�п��õ�32λ�Ĵ��������� " << devProp.regsPerBlock << std::endl;
		std::cout << "======================================================" << std::endl;
	}
}

//�������
void testoperation(){
	srand(unsigned(time(NULL)));
	// ����״̬����
	hipblasStatus_t status;
	const int N = 10, M = 5;
 
	// �� "�ڴ�" ��Ϊ��Ҫ����ľ��󿪱ٿռ�
	float *h_A = (float*)malloc(N * M * sizeof(float));
	float *h_B = (float*)malloc(N * M * sizeof(float));
	// �� "�ڴ�" ��Ϊ��Ҫ����������ľ��󿪱ٿռ�
	float *h_C = (float*)malloc(M * M * sizeof(float));
 
	// Ϊ����������Ԫ�ظ��� 0-10 ��Χ�ڵ������
	for (int i = 0; i < N * M; i++) {
		h_A[i] = (float)(rand() % 10 + 1);
		h_B[i] = (float)(rand() % 10 + 1);
	}
 
	// ��ӡ�����Եľ���
	std::cout << "���� A :" << std::endl;
	for (int i = 0; i < N * M; i++){
		std::cout << h_A[i] << " ";
		if ((i + 1) % N == 0) std::cout << std::endl;
	}
	std::cout << std::endl;

	std::cout << "���� B :" << std::endl;
	for (int i = 0; i < N * M; i++){
		std::cout << h_B[i] << " ";
		if ((i + 1) % M == 0) std::cout << std::endl;
	}
	std::cout << std::endl;

	// GPU ����������

	// ��������ʼ�� CUBLAS �����
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
 
	if (status != HIPBLAS_STATUS_SUCCESS){
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED){
			std::cout << "CUBLAS ����ʵ��������" << std::endl;
		}
		return;
	}
 
	float *d_A, *d_B, *d_C;
	// �� "�Դ�" ��Ϊ��Ҫ����ľ��󿪱ٿռ�
	hipMalloc(
		(void**)&d_A,    // ָ�򿪱ٵĿռ��ָ��
		N*M * sizeof(float)    //����Ҫ���ٿռ���ֽ���
		);
	hipMalloc((void**)&d_B, N * M * sizeof(float));
	// �� "�Դ�" ��Ϊ��Ҫ����������ľ��󿪱ٿռ�
	hipMalloc((void**)&d_C, M * M * sizeof(float));
 
	// ���������ݴ��ݽ� �Դ� ���Ѿ����ٺ��˵Ŀռ�
	hipblasSetVector(
		N * M,    // Ҫ�����Դ��Ԫ�ظ���
		sizeof(float),    // ÿ��Ԫ�ش�С
		h_A,    // ��������ʼ��ַ
		1,      // ����Ԫ��֮��Ĵ洢���
		d_A,    // GPU ����ʼ��ַ
		1       // ����Ԫ��֮��Ĵ洢���
		);
	hipblasSetVector(N * M, sizeof(float), h_B, 1, d_B, 1);
 
	// ͬ������
	hipDeviceSynchronize();
 
	// ���ݽ�������˺����еĲ��������庬����ο������ֲᡣ
	float a = 1; float b = 0;
	// ������ˡ��ú�����Ȼ���������������������
	hipblasSgemm(
		handle,    // blas ����� 
		HIPBLAS_OP_T,    // ���� A ���Բ���
		HIPBLAS_OP_T,    // ���� B ���Բ���
		M,      // A, C ������ 
		M,      // B, C ������
		N,      // A �������� B ������
		&a,     // ����ʽ�� �� ֵ
		d_A,    // A ���Դ��еĵ�ַ
		N,      // lda
		d_B,    // B ���Դ��еĵ�ַ
		M,      // ldb
		&b,     // ����ʽ�� �� ֵ
		d_C,    // C ���Դ��еĵ�ַ(�������)
		M       // ldc
		);
 
	// ͬ������
	hipDeviceSynchronize();
	
	// �� �Դ� ��ȡ���������� �ڴ���ȥ
	hipblasGetVector(
		M*M,    //  Ҫȡ��Ԫ�صĸ���
		sizeof(float),    // ÿ��Ԫ�ش�С
		d_C,    // GPU ����ʼ��ַ
		1,      // ����Ԫ��֮��Ĵ洢���
		h_C,    // ��������ʼ��ַ
		1       // ����Ԫ��֮��Ĵ洢���
		);
 
	// ��ӡ������
	std::cout << "��������ת�� ( (A*B)��ת�� )��" << std::endl;
 
	for (int i = 0; i<M*M; i++){
		std::cout << h_C[i] << " ";
		if ((i + 1) % M == 0) std::cout << std::endl;
	}
 
	// �����ʹ�ù����ڴ�
	free(h_A); free(h_B); free(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	// �ͷ� CUBLAS �����
	hipblasDestroy(handle);
}
